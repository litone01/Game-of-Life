#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <ctype.h>
#include <errno.h>
#include "exporter.h"
#include "settings.h"

// including the "dead faction": 0
#define MAX_FACTIONS 10

// this macro is here to make the code slightly more readable, not because it can be safely changed to
// any integer value; changing this to a non-zero value may break the code
#define DEAD_FACTION 0

// death toll due to fighting
__device__ __managed__ int deathToll;


void check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
    }

}

/**
 * Specifies the number(s) of live neighbors of the same faction required for a dead cell to become alive.
 */
__device__ bool isBirthable(int n)
{
    return n == 3;
}

/**
 * Specifies the number(s) of live neighbors of the same faction required for a live cell to remain alive.
 */
__device__ bool isSurvivable(int n)
{
    return n == 2 || n == 3;
}

/**
 * Specifies the number of live neighbors of a different faction required for a live cell to die due to fighting.
 */
__device__ bool willFight(int n) {
    return n > 0;
}

/**
 * returns the value at the input row and col of the input grid, if valid.
 * 
 * -1 is returned if row or col is out of bounds (as specified by nRows and nCols).
 */
__device__ int getValueAtCuda(const int *grid, int nRows, int nCols, int row, int col)
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols)
    {
        return -1;
    }

    return *(grid + (row * nCols) + col);
}

/**
 * sets the value at the input row and col of the input grid to val.
 * 
 * Does nothing if row or col is out of bounds (as specified by nRows and nCols).
 */
__device__ void setValueAtCuda(int *grid, int nRows, int nCols, int row, int col, int val)
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols)
    {
        return;
    }

    *(grid + (row * nCols) + col) = val;
}


/**
 * returns the value at the input row and col of the input grid, if valid.
 * 
 * -1 is returned if row or col is out of bounds (as specified by nRows and nCols).
 */
int getValueAt(const int *grid, int nRows, int nCols, int row, int col)
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols)
    {
        return -1;
    }

    return *(grid + (row * nCols) + col);
}

/**
 * sets the value at the input row and col of the input grid to val.
 * 
 * Does nothing if row or col is out of bounds (as specified by nRows and nCols).
 */
void setValueAt(int *grid, int nRows, int nCols, int row, int col, int val)
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols)
    {
        return;
    }

    *(grid + (row * nCols) + col) = val;
}

/**
 * Writes the input world to stdout.
 */
void printWorld(const int *world, int nRows, int nCols)
{
    for (int row = 0; row < nRows; row++)
    {
        for (int col = 0; col < nCols; col++)
        {
            printf("%d ", *(world + (row * nCols) + col));
        }
        printf("\n");
    }
}


/**
 * Computes and returns the next state of the cell specified by row and col based on currWorld and invaders. Sets *diedDueToFighting to
 * true if this cell should count towards the death toll due to fighting.
 * 
 * invaders can be NULL if there are no invaders.
 */
__device__ int getNextState(const int *currWorld, const int *invaders, int nRows, int nCols, int row, int col, bool *diedDueToFighting)
{
    // we'll explicitly set if it was death due to fighting
    *diedDueToFighting = false;

    // faction of this cell
    int cellFaction = getValueAtCuda(currWorld, nRows, nCols, row, col);

    // did someone just get landed on?
    if (invaders != NULL && getValueAtCuda(invaders, nRows, nCols, row, col) != DEAD_FACTION)
    {
        *diedDueToFighting = cellFaction != DEAD_FACTION;
        return getValueAtCuda(invaders, nRows, nCols, row, col);
    }

    // tracks count of each faction adjacent to this cell
    int neighborCounts[MAX_FACTIONS];
    memset(neighborCounts, 0, MAX_FACTIONS * sizeof(int));

    // count neighbors (and self)
    for (int dy = -1; dy <= 1; dy++)
    {
        for (int dx = -1; dx <= 1; dx++)
        {
            int faction = getValueAtCuda(currWorld, nRows, nCols, row + dy, col + dx);
            if (faction >= DEAD_FACTION)
            {
                neighborCounts[faction]++;
            }
        }
    }

    // we counted this cell as its "neighbor"; adjust for this
    neighborCounts[cellFaction]--;

    if (cellFaction == DEAD_FACTION)
    {
        // this is a dead cell; we need to see if a birth is possible:
        // need exactly 3 of a single faction; we don't care about other factions

        // by default, no birth
        int newFaction = DEAD_FACTION;

        // start at 1 because we ignore dead neighbors
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++)
        {
            int count = neighborCounts[faction];
            if (isBirthable(count))
            {
                newFaction = faction;
            }
        }

        return newFaction;
    }
    else
    {
        /** 
         * this is a live cell; we follow the usual rules:
         * Death (fighting): > 0 hostile neighbor
         * Death (underpopulation): < 2 friendly neighbors and 0 hostile neighbors
         * Death (overpopulation): > 3 friendly neighbors and 0 hostile neighbors
         * Survival: 2 or 3 friendly neighbors and 0 hostile neighbors
         */

        int hostileCount = 0;
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++)
        {
            if (faction == cellFaction)
            {
                continue;
            }
            hostileCount += neighborCounts[faction];
        }

        if (willFight(hostileCount))
        {
            *diedDueToFighting = true;
            return DEAD_FACTION;
        }

        int friendlyCount = neighborCounts[cellFaction];
        if (!isSurvivable(friendlyCount))
        {
            return DEAD_FACTION;
        }

        return cellFaction;
    }
}


__global__ void simulate(int *wholeNewWorld, const int *world, const int * inv, int nRows, int nCols, int elementPerThread)
{
    // printf("Currently on block %d %d %d, thread %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);

    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    
    // calculate the starting and ending position of the world to be computed by the current thread
    int firstElement = threadId * elementPerThread;
    int lastElement = ((threadId + 1) * elementPerThread) - 1;
    // handle cases where nRows * nCols are not divisible by nThreads.
    int max = nRows * nCols - 1;
    if (lastElement > max) {
        lastElement = max;
    }

    int currentElement = firstElement;
    int row, col;
    while (currentElement <= lastElement) {
        row = currentElement / nCols;
        col = currentElement % nCols;
        // printf("[thread %d] row:%d, col:%d\n", threadId, row, col);
        bool diedDueToFighting;
        int nextState = getNextState(world, inv, nRows, nCols, row, col, &diedDueToFighting);
        setValueAtCuda(wholeNewWorld, nRows, nCols, row, col, nextState);
        if (diedDueToFighting) {
            // critical region
            atomicAdd(&deathToll, 1);
        }
        currentElement++;
    }
}



/**
 * The main simulation logic.
 * 
 * goi does not own startWorld, invasionTimes or invasionPlans and should not modify or attempt to free them.
 * nThreads is the number of threads to simulate with. It is ignored by the sequential implementation.
 */
int goi(int nGenerations, const int *startWorld, int nRows, int nCols, int nInvasions, const int *invasionTimes, int **invasionPlans,
        int gridX, int gridY, int gridZ, int blockX, int blockY, int blockZ)
{
    hipError_t rc;
    size_t worldSizeInBytes = nRows * nCols * sizeof(int);

    // grid and block dimensions
    dim3 gridDim(gridX, gridY, gridZ);
    dim3 blockDim(blockX, blockY, blockZ);
    int nThreads = blockX * blockY * blockZ * gridX * gridY * gridZ;
    // rounding up the division to the nearest integer
    // further assume that nRows * nCols >= nThreads
    int elementPerThread = 1 +((nRows * nCols - 1) / nThreads);

    // init deathtoll
    deathToll = 0;
    // rc = hipMemcpyToSymbol(HIP_SYMBOL(deathToll), res, sizeof(res));
    // if (rc != hipSuccess)
    // {
    //     printf("Could not copy to device. Reason: %s\n", hipGetErrorString(rc));
    // }

    // init the world!
    // we make a copy because we do not own startWorld (and will perform free() on hostWorld)
    int* world = (int*) malloc(worldSizeInBytes);
    if (world == NULL)
    {
        return -1;
    }

    // set value of the host world
    // TODO (maybe can just use the startWorld since we are not modifying)
    for (int row = 0; row < nRows; row++)
    {
        for (int col = 0; col < nCols; col++)
        {
            setValueAt(world, nRows, nCols, row, col, getValueAt(startWorld, nRows, nCols, row, col));
        }
    }

    // Copy world to CUDA
    int* world_cuda;
    hipMalloc((void **)&world_cuda, sizeof(int) * nRows * nCols);
    rc = hipMemcpy(world_cuda, world, sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);
    if (rc != hipSuccess)
    {
        printf("Could not copy world_cuda to device. Reason: %s\n", hipGetErrorString(rc));
    }

#if PRINT_GENERATIONS
    printf("\n=== WORLD 0 ===\n");
    printWorld(world, nRows, nCols);
#endif

#if EXPORT_GENERATIONS
    exportWorld(world, nRows, nCols);
#endif

    // Begin simulating
    int invasionIndex = 0;
    for (int i = 1; i <= nGenerations; i++)
    {
        // is there an invasion this generation?
        int *inv = NULL;
        int *inv_cuda = NULL;
        if (invasionIndex < nInvasions && i == invasionTimes[invasionIndex])
        {
            // we make a copy because we do not own invasionPlans
            inv = (int*) malloc(worldSizeInBytes);
            if (inv == NULL)
            {
                free(world);
                hipFree(world_cuda);
                return -1;
            }
            rc = hipMalloc((void**) &inv_cuda, worldSizeInBytes);
            if (rc != hipSuccess) {
                free(world);
                hipFree(world_cuda);
                free(inv);
                return -1;
            }

            for (int row = 0; row < nRows; row++)
            {
                for (int col = 0; col < nCols; col++)
                {
                    setValueAt(inv, nRows, nCols, row, col, getValueAt(invasionPlans[invasionIndex], nRows, nCols, row, col));
                }
            }
            invasionIndex++;
            hipMemcpy(inv_cuda, inv, worldSizeInBytes, hipMemcpyHostToDevice);
        }

        // create newWorld state for cuda
        int *wholeNewWorld_cuda;
        rc = hipMalloc((void **)&wholeNewWorld_cuda, sizeof(int) * nRows * nCols);
        if (rc != hipSuccess) {
                free(world);
                hipFree(world_cuda);
                free(inv);
                hipFree(inv_cuda);
                return -1;
        }

        // Simulate next state
        simulate<<<gridDim, blockDim>>>(wholeNewWorld_cuda, world_cuda, inv_cuda, nRows, nCols, elementPerThread);
        check_cuda_errors();
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess) {
            printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        }

        // free inv and inv_cuda if needed
        if (inv != NULL)
        {
            free(inv);
            hipFree(inv_cuda);
        }

        // swap worlds
        hipFree(world_cuda);
        world_cuda = wholeNewWorld_cuda;


#if PRINT_GENERATIONS
        hipMemcpy(world, world_cuda, worldSizeInBytes, hipMemcpyDeviceToHost);
        printf("\n=== WORLD %d ===\n", i);
        printWorld(world, nRows, nCols);
#endif

#if EXPORT_GENERATIONS
        hipMemcpy(world, world_cuda, worldSizeInBytes, hipMemcpyDeviceToHost);
        exportWorld(world, nRows, nCols);
#endif
    }

    free(world);
    hipFree(world_cuda);

    return deathToll;
}


